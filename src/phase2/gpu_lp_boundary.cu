#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#ifdef function
#undef function
#endif
#include <vector>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <utility>
#include <unordered_set>
#include <unordered_map>
#include <algorithm>
#include <cstring>
#include <omp.h>
#include <chrono>

#include "phase2/gpu_lp_boundary.h"

// ==================== 바운더리 서브그래프 생성 (로컬+고스트 통합) ====================

/**
 * 바운더리 노드 + 1-hop 이웃으로 구성된 통합 서브그래프 생성 (최적화)
 * - 로컬 노드와 고스트 노드의 라벨을 통합하여 GPU에 전달
 * - 로컬 노드만 업데이트 대상으로 표시
 * - GPU 메모리 지역성 최적화 + OpenMP 병렬화
 */
BoundarySubgraph createBoundarySubgraphUnified(
    const std::vector<int>& row_ptr,
    const std::vector<int>& col_idx,
    const std::vector<int>& boundary_nodes,
    const std::vector<int>& local_labels,
    const std::vector<int>& ghost_labels,
    const std::vector<int>& global_ids,
    int num_local_nodes)
{
    BoundarySubgraph subgraph;
    subgraph.num_local_nodes = num_local_nodes;
    int total_nodes = local_labels.size() + ghost_labels.size();
    
    // 1단계: 서브그래프에 포함될 모든 노드 수집 (바운더리 + 1-hop 이웃) - 병렬화
    std::vector<bool> node_included(total_nodes, false);
    
    // 바운더리 노드들 먼저 마킹
    #pragma omp parallel for schedule(static)
    for (size_t i = 0; i < boundary_nodes.size(); i++) {
        int boundary_node = boundary_nodes[i];
        if (boundary_node >= 0 && boundary_node < total_nodes) {
            node_included[boundary_node] = true;
        }
    }
    
    // 각 바운더리 노드의 1-hop 이웃 추가 - 병렬화
    #pragma omp parallel for schedule(dynamic, 10)
    for (size_t i = 0; i < boundary_nodes.size(); i++) {
        int boundary_node = boundary_nodes[i];
        if (boundary_node >= 0 && boundary_node < (int)row_ptr.size() - 1) {
            for (int edge_idx = row_ptr[boundary_node]; edge_idx < row_ptr[boundary_node + 1]; edge_idx++) {
                int neighbor = col_idx[edge_idx];
                if (neighbor >= 0 && neighbor < total_nodes) {
                    node_included[neighbor] = true;
                }
            }
        }
    }
    
    // 2단계: 포함된 노드들을 벡터로 변환 (병렬 압축)
    std::vector<int> subgraph_nodes;
    
    // 더 정확한 예약: 바운더리 노드 수 + 평균 degree 추정
    size_t estimated_size = boundary_nodes.size() * 2; // 바운더리 + 1-hop 이웃 추정
    subgraph_nodes.reserve(std::min(estimated_size, static_cast<size_t>(total_nodes)));
    
    for (int i = 0; i < total_nodes; i++) {
        if (node_included[i]) {
            subgraph_nodes.push_back(i);
        }
    }
    
    subgraph.num_nodes = subgraph_nodes.size();
    subgraph.node_mapping = std::move(subgraph_nodes);
    subgraph.reverse_mapping.assign(total_nodes, -1);
    
    // 통합 라벨 배열 및 로컬 노드 플래그 구성 - 병렬화
    subgraph.labels.resize(subgraph.num_nodes);
    subgraph.local_node_flags.resize(subgraph.num_nodes);
    
    #pragma omp parallel for
    for (int i = 0; i < subgraph.num_nodes; i++) {
        int orig_node = subgraph.node_mapping[i];
        subgraph.reverse_mapping[orig_node] = i;
        
        // 라벨 설정 (로컬 또는 고스트)
        if (orig_node < num_local_nodes) {
            // 로컬 노드
            subgraph.labels[i] = local_labels[orig_node];
            subgraph.local_node_flags[i] = 1;
        } else {
            // 고스트 노드
            int ghost_idx = orig_node - num_local_nodes;
            if (ghost_idx >= 0 && ghost_idx < (int)ghost_labels.size()) {
                subgraph.labels[i] = ghost_labels[ghost_idx];
            } else {
                // 고스트 노드가 유효하지 않은 경우
                printf("Warning: Invalid ghost node index %d for original node %d\n", ghost_idx, orig_node);
                exit(1);
                //subgraph.labels[i] = -1; // 유효하지 않은 라벨
            }
            subgraph.local_node_flags[i] = 0;
        }
    }
    
    // 3단계: 서브그래프 CSR 구성 - 메모리 효율적 방식
    subgraph.row_ptr.resize(subgraph.num_nodes + 1, 0);
    
    // 먼저 각 노드의 이웃 수 계산 (병렬 + 캐시 친화적)
    std::vector<int> neighbor_counts(subgraph.num_nodes, 0);
    
    #pragma omp parallel for schedule(static, 64) // 캐시 라인 크기 고려
    for (int i = 0; i < subgraph.num_nodes; i++) {
        int orig_node = subgraph.node_mapping[i];
        if (orig_node < (int)row_ptr.size() - 1) {
            for (int edge_idx = row_ptr[orig_node]; edge_idx < row_ptr[orig_node + 1]; edge_idx++) {
                int neighbor = col_idx[edge_idx];
                if (neighbor >= 0 && neighbor < total_nodes && subgraph.reverse_mapping[neighbor] != -1) {
                    neighbor_counts[i]++;
                }
            }
        }
    }
    
    // CSR row_ptr 계산 (prefix sum)
    int edge_count = 0;
    for (int i = 0; i < subgraph.num_nodes; i++) {
        subgraph.row_ptr[i] = edge_count;
        edge_count += neighbor_counts[i];
    }
    subgraph.row_ptr[subgraph.num_nodes] = edge_count;
    subgraph.num_edges = edge_count;
    
    // 이웃 노드 수집 (병렬 + 메모리 지역성 최적화)
    subgraph.col_idx.resize(edge_count);
    
    #pragma omp parallel for schedule(static, 32)
    for (int i = 0; i < subgraph.num_nodes; i++) {
        int orig_node = subgraph.node_mapping[i];
        int start_idx = subgraph.row_ptr[i];
        int idx = start_idx;
        
        if (orig_node < (int)row_ptr.size() - 1) {
            for (int edge_idx = row_ptr[orig_node]; edge_idx < row_ptr[orig_node + 1]; edge_idx++) {
                int neighbor = col_idx[edge_idx];
                if (neighbor >= 0 && neighbor < total_nodes) {
                    int neighbor_subgraph_idx = subgraph.reverse_mapping[neighbor];
                    if (neighbor_subgraph_idx != -1) {
                        subgraph.col_idx[idx++] = neighbor_subgraph_idx;
                    }
                }
            }
        }
    }
    
    // 4단계: 서브그래프 내 실제 바운더리 노드 인덱스 찾기 (로컬 노드만) - 최적화
    std::vector<bool> is_boundary(boundary_nodes.size(), true);
    std::vector<int> boundary_subgraph_indices;
    boundary_subgraph_indices.reserve(boundary_nodes.size());
    
    for (int boundary_node : boundary_nodes) {
        if (boundary_node < num_local_nodes) { // 로컬 노드만
            int subgraph_idx = subgraph.reverse_mapping[boundary_node];
            if (subgraph_idx != -1) {
                boundary_subgraph_indices.push_back(subgraph_idx);
            }
        }
    }
    
    subgraph.boundary_indices = std::move(boundary_subgraph_indices);
    
    return subgraph;
}

/**
 * 적응적 바운더리 확장 (메모리 및 성능 최적화 버전)
 * - std::unordered_set 대신 'Sort-Unique' 방식으로 메모리 사용량 최소화
 * - OpenMP 병렬 처리 효율성 극대화
 */
std::vector<int> expandBoundaryNodes(
    const std::vector<int>& row_ptr,
    const std::vector<int>& col_idx,
    const std::vector<int>& prev_boundary_nodes,
    const std::vector<int>& labels,
    const std::vector<double>& penalty,
    const std::vector<double>& RE,
    int vertex_count,
    int iter)
{
    // --- 1단계: 후보 노드 병렬 수집 ---
    // 각 OpenMP 스레드가 자신만의 로컬 벡터에 후보 노드를 수집하여 메모리 경합을 방지합니다.
    std::vector<std::vector<int>> local_candidates_per_thread;
    
    #pragma omp parallel
    {
        int thread_id = omp_get_thread_num();
        #pragma omp single
        {
            local_candidates_per_thread.resize(omp_get_num_threads());
        }

        #pragma omp for schedule(dynamic, 100) nowait
        for (size_t i = 0; i < prev_boundary_nodes.size(); ++i) {
            int boundary_node = prev_boundary_nodes[i];
            if (boundary_node >= 0 && boundary_node < vertex_count) {
                local_candidates_per_thread[thread_id].push_back(boundary_node);
                if (boundary_node < (int)row_ptr.size() - 1) {
                    for (int edge_idx = row_ptr[boundary_node]; edge_idx < row_ptr[boundary_node + 1]; ++edge_idx) {
                        int neighbor = col_idx[edge_idx];
                        if (neighbor >= 0 && neighbor < vertex_count) {
                            local_candidates_per_thread[thread_id].push_back(neighbor);
                        }
                    }
                }
            }
        }
    }

    // --- 2단계: 모든 후보 노드 병합 ---
    // 각 스레드가 수집한 로컬 벡터들을 하나의 큰 벡터로 합칩니다.
    std::vector<int> all_candidates;
    size_t total_candidates = 0;
    for (const auto& vec : local_candidates_per_thread) {
        total_candidates += vec.size();
    }
    all_candidates.reserve(total_candidates);
    for (const auto& vec : local_candidates_per_thread) {
        all_candidates.insert(all_candidates.end(), vec.begin(), vec.end());
    }
    local_candidates_per_thread.clear(); // 사용한 메모리는 즉시 해제합니다.

    // --- 3단계: 정렬 및 중복 제거 (Sort-Unique) ---
    // std::unordered_set 대신 정렬 후 중복을 제거하는 방식으로 메모리 효율성을 극대화합니다.
    std::sort(all_candidates.begin(), all_candidates.end());
    all_candidates.erase(std::unique(all_candidates.begin(), all_candidates.end()), all_candidates.end());

    // --- 4단계: 실제 경계 노드 필터링 및 선택 (기존 로직과 동일) ---
    
    double ratio = exp((-iter)/(5.0)); // 분모 값을 조절하여 선택 비율을 변경할 수 있습니다.
    printf("===========>>> ratio : %f <<<============", ratio);

    std::unordered_map<int, std::vector<std::pair<int,int>>> part_to_nodes;
    part_to_nodes.reserve(64);

    // 이제 중복이 제거된 후보 노드들로 실제 경계 노드를 찾습니다.
    for (int node : all_candidates) {
        if (node + 1 >= static_cast<int>(row_ptr.size())) continue;

        const int node_label = labels[node];
        bool is_boundary = false;

        for (int e = row_ptr[node]; e < row_ptr[node + 1]; ++e) {
            int nbr = col_idx[e];
            if (nbr < 0 || nbr >= vertex_count) continue;
            if (labels[nbr] != node_label) {
                is_boundary = true;
                break;
            }
        }

        if (is_boundary) {
            int degree = row_ptr[node + 1] - row_ptr[node];
            part_to_nodes[node_label].emplace_back(node, degree);
        }
    }

    std::vector<int> selected_boundary_nodes;
    size_t approx_total = 0;
    for (const auto& kv : part_to_nodes) approx_total += kv.second.size();
    selected_boundary_nodes.reserve(static_cast<size_t>(std::ceil(ratio * approx_total)) + 8);

    for (auto& kv : part_to_nodes) {
        int part_id = kv.first;
        auto& vec = kv.second;

        // 불균형 상태(RE)에 따라 정렬 기준을 다르게 하여 노드를 선택합니다.
        if (RE[part_id] > 1.0) {
            std::sort(vec.begin(), vec.end(),
                      [](const auto& a, const auto& b){
                          if (a.second != b.second) return a.second > b.second; // 차수(degree) 내림차순
                          return a.first  < b.first;
                      });
        } else {
            std::sort(vec.begin(), vec.end(),
                      [](const auto& a, const auto& b){
                          if (a.second != b.second) return a.second < b.second; // 차수(degree) 오름차순
                          return a.first  < b.first;
                      });
        }

        int k = static_cast<int>(std::ceil(ratio * static_cast<double>(vec.size())));
        if (k > static_cast<int>(vec.size())) k = static_cast<int>(vec.size());

        for (int i = 0; i < k; ++i) {
            selected_boundary_nodes.push_back(vec[i].first);
        }
    }
    
    return selected_boundary_nodes;
}

// ==================== 단순화된 GPU 메모리 관리 ====================
class GPUMemoryManager {
public:
    static hipError_t safeMalloc(void** ptr, size_t size) {
        return hipMalloc(ptr, size);
    }
    
    static hipError_t safeFree(void* ptr) {
        return hipFree(ptr);
    }
};

// ==================== 서브그래프 전용 커널 (로컬 노드만 업데이트) ====================

/**
 * 통합 서브그래프 전용 최적화 커널 (워프 기반 처리)
 * - 각 워프(32개 스레드)가 하나의 바운더리 노드를 협력 처리
 * - 워프 내 스레드들이 이웃 노드들을 병렬로 처리
 * - 큰 degree 노드에서 성능 향상
 */
__global__ void boundaryLPKernel_unified(
    const int* __restrict__ row_ptr, 
    const int* __restrict__ col_idx,
    const int* __restrict__ labels_old, 
    int* __restrict__ labels_new,
    const int* __restrict__ local_node_flags,
    const double* __restrict__ penalty,
    const int* __restrict__ boundary_indices, 
    int boundary_count,
    int num_partitions,
    int subgraph_size)
{
    // 워프 관련 정보
    const int WARP_SIZE = 32;
    int warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / WARP_SIZE;
    int lane_id = threadIdx.x % WARP_SIZE;
    int warp_in_block = threadIdx.x / WARP_SIZE;
    
    // 워프가 담당할 바운더리 노드 결정
    if (warp_id >= boundary_count) return;
    
    int subgraph_node_idx = boundary_indices[warp_id];
    if (subgraph_node_idx < 0 || subgraph_node_idx >= subgraph_size) return;
    
    // 로컬 노드가 아니면 조기 종료 (고스트 노드는 업데이트하지 않음)
    if (local_node_flags[subgraph_node_idx] != 1) return;
    
    int my_label = labels_old[subgraph_node_idx];
    
    // 워프별 공유 메모리: 각 워프가 독립적인 스코어 배열 사용
    // 블록당 최대 8개 워프 (256/32), 최대 32개 파티션
    __shared__ double warp_scores[8][32];
    
    int effective_partitions = min(num_partitions, 32);
    
    // 스코어 배열 초기화: 각 스레드가 담당 파티션을 0으로 설정
    if (lane_id < effective_partitions) {
        warp_scores[warp_in_block][lane_id] = 0.0;
    }
    __syncwarp(); // 워프 내 모든 스레드 동기화
    
    // 이웃 노드 처리: stride 방식으로 병렬 순회
    int start = row_ptr[subgraph_node_idx];
    int end = row_ptr[subgraph_node_idx + 1];
    
    // 각 스레드가 stride=32로 이웃들을 처리
    // Thread 0: edge 0, 32, 64, ...
    // Thread 1: edge 1, 33, 65, ...
    for (int e = start + lane_id; e < end; e += WARP_SIZE) {
        int neighbor_idx = col_idx[e];
        if (neighbor_idx >= 0 && neighbor_idx < subgraph_size) {
            int neighbor_label = labels_old[neighbor_idx];
            if (neighbor_label >= 0 && neighbor_label < effective_partitions) {
                // atomic 연산으로 스레드 간 경쟁 상태 방지
                atomicAdd(&warp_scores[warp_in_block][neighbor_label], 1.0);
            }
        }
    }
    __syncwarp(); // 모든 이웃 처리 완료까지 대기
    
    // 워프 대표 스레드(lane 0)가 최종 라벨 결정
    if (lane_id == 0) {
        // 1단계: 패널티 적용하여 최종 스코어 계산
        for (int l = 0; l < effective_partitions; l++) {
            if (warp_scores[warp_in_block][l] > 0.0) {
                warp_scores[warp_in_block][l] = warp_scores[warp_in_block][l] * (1.0 + penalty[l]);
            }
        }
        
        // 2단계: 최고 스코어를 가진 라벨 찾기
        int best_label = my_label;
        double best_score = (my_label >= 0 && my_label < effective_partitions) ? 
                           warp_scores[warp_in_block][my_label] : 0.0;
        
        for (int l = 0; l < effective_partitions; l++) {
            if (warp_scores[warp_in_block][l] > best_score) {
                best_score = warp_scores[warp_in_block][l];
                best_label = l;
            }
        }
        
        // 3단계: 새 라벨 저장
        labels_new[subgraph_node_idx] = best_label;
    }
}

// ==================== GPU 처리 함수 ====================

/**
 * GPU 기반 바운더리 라벨 전파 - 통합 서브그래프 처리
 * 
 * 특징:
 * - 바운더리 서브그래프 기반으로 메모리 효율성 극대화
 * - 워프 기반 병렬 처리로 GPU 성능 최적화
 * - 로컬 노드만 업데이트하여 MPI 일관성 보장
 * - 스트리밍 방식으로 대용량 그래프 처리 지원
 */
GPULabelUpdateResult runBoundaryLPOnGPU_SubgraphUnified(
    const BoundarySubgraph& subgraph,
    const std::vector<double>& penalty,
    int num_partitions)
{
    GPULabelUpdateResult result;
    
    // 비동기 처리를 위한 CUDA 스트림
    hipStream_t stream;
    hipStreamCreate(&stream);
    
    // GPU 메모리 할당 (서브그래프 크기만)
    int *d_row_ptr, *d_col_idx, *d_labels_old, *d_labels_new;
    int *d_local_flags, *d_boundary_indices;
    double *d_penalty;
    
    GPUMemoryManager::safeMalloc((void**)&d_row_ptr, subgraph.row_ptr.size() * sizeof(int));
    GPUMemoryManager::safeMalloc((void**)&d_col_idx, subgraph.col_idx.size() * sizeof(int));
    GPUMemoryManager::safeMalloc((void**)&d_labels_old, subgraph.num_nodes * sizeof(int));
    GPUMemoryManager::safeMalloc((void**)&d_labels_new, subgraph.num_nodes * sizeof(int));
    GPUMemoryManager::safeMalloc((void**)&d_local_flags, subgraph.num_nodes * sizeof(int));
    GPUMemoryManager::safeMalloc((void**)&d_boundary_indices, subgraph.boundary_indices.size() * sizeof(int));
    GPUMemoryManager::safeMalloc((void**)&d_penalty, penalty.size() * sizeof(double));
    
    // 비동기 메모리 전송
    hipMemcpyAsync(d_row_ptr, subgraph.row_ptr.data(), subgraph.row_ptr.size() * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_col_idx, subgraph.col_idx.data(), subgraph.col_idx.size() * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_labels_old, subgraph.labels.data(), subgraph.num_nodes * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_labels_new, subgraph.labels.data(), subgraph.num_nodes * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_local_flags, subgraph.local_node_flags.data(), subgraph.num_nodes * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_boundary_indices, subgraph.boundary_indices.data(), subgraph.boundary_indices.size() * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_penalty, penalty.data(), penalty.size() * sizeof(double), hipMemcpyHostToDevice, stream);
    
    // 커널 실행 설정 (워프 기반 - 각 워프가 하나의 노드 담당)
    const int WARP_SIZE = 32;
    int boundary_count = subgraph.boundary_indices.size();
    int total_warps = boundary_count; // 각 워프가 하나의 바운더리 노드 담당
    int threads_per_block = 256; // 블록당 256개 스레드 (8개 워프)
    int warps_per_block = threads_per_block / WARP_SIZE;
    int blocks = (total_warps + warps_per_block - 1) / warps_per_block;
    
    // 공유 메모리 계산: 워프별 파티션 배열 (8워프 x 32파티션)
    size_t shared_mem = 8 * 32 * sizeof(double); // 고정 크기
    
    boundaryLPKernel_unified<<<blocks, threads_per_block, shared_mem, stream>>>(
        d_row_ptr, d_col_idx, d_labels_old, d_labels_new, d_local_flags, d_penalty,
        d_boundary_indices, subgraph.boundary_indices.size(),
        num_partitions, subgraph.num_nodes);
    
    // GPU 커널 실행 완료 대기
    hipStreamSynchronize(stream);
    
    // 결과 복사
    std::vector<int> updated_labels(subgraph.num_nodes);
    hipMemcpyAsync(updated_labels.data(), d_labels_new, subgraph.num_nodes * sizeof(int), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    
    // 변경된 로컬 노드만 추출
    for (int i = 0; i < subgraph.num_nodes; i++) {
        if (subgraph.local_node_flags[i] == 1) { // 로컬 노드만
            if (subgraph.labels[i] != updated_labels[i]) { // 라벨이 변경된 경우
                int orig_node_id = subgraph.node_mapping[i];
                result.updated_nodes.push_back(orig_node_id);
                result.updated_labels.push_back(updated_labels[i]);
                result.change_count++;
            }
        }
    }
    
    // 리소스 정리
    hipStreamDestroy(stream);
    GPUMemoryManager::safeFree(d_row_ptr);
    GPUMemoryManager::safeFree(d_col_idx);
    GPUMemoryManager::safeFree(d_labels_old);
    GPUMemoryManager::safeFree(d_labels_new);
    GPUMemoryManager::safeFree(d_local_flags);
    GPUMemoryManager::safeFree(d_boundary_indices);
    GPUMemoryManager::safeFree(d_penalty);
    
    return result;
}

/**
 * 스트리밍 방식 GPU 처리 (개선된 인터페이스)
 */
GPULabelUpdateResult runBoundaryLPOnGPU_Streaming(
    const std::vector<int>& row_ptr,
    const std::vector<int>& col_idx,
    const std::vector<int>& boundary_nodes,
    const std::vector<int>& local_labels,
    const std::vector<int>& ghost_labels,
    const std::vector<int>& global_ids,
    const std::vector<double>& penalty,
    int num_local_nodes,
    int num_partitions,
    size_t max_memory_mb)
{
    // 통합 서브그래프 생성
    BoundarySubgraph subgraph = createBoundarySubgraphUnified(
        row_ptr, col_idx, boundary_nodes, local_labels, ghost_labels, global_ids, num_local_nodes);
    printf("Subgraph created with %d nodes and %d edges\n", subgraph.num_nodes, subgraph.num_edges);
    // 메모리 사용량 계산 (수정된 최종 버전)
    size_t row_ptr_bytes = subgraph.row_ptr.size() * sizeof(int);
    size_t col_idx_bytes = subgraph.col_idx.size() * sizeof(int);
    size_t labels_bytes = subgraph.labels.size() * sizeof(int);
    size_t flags_bytes = subgraph.local_node_flags.size() * sizeof(int);
    size_t boundary_bytes = subgraph.boundary_indices.size() * sizeof(int);
    size_t penalty_bytes = penalty.size() * sizeof(double);

    size_t subgraph_memory = row_ptr_bytes +
                            col_idx_bytes +
                            (labels_bytes * 2) + // d_labels_old 와 d_labels_new, 총 2개
                            flags_bytes +
                            boundary_bytes +
                            penalty_bytes;

    size_t available_memory = max_memory_mb * 1024 * 1024;
    
    if (subgraph_memory <= available_memory) {
        // 전체 서브그래프가 메모리에 들어가는 경우
        return runBoundaryLPOnGPU_SubgraphUnified(subgraph, penalty, num_partitions);
    } else {
        // 청크 단위 처리 필요
        return runBoundaryLPOnGPU_Chunked(subgraph, penalty, num_partitions, available_memory);
    }
}

/**
 * 청크 단위 처리 함수 구현 (수정된 최종 버전)
 * - 큰 서브그래프를 메모리에 맞게 청크로 나누어 순차적으로 처리하고 결과를 병합
 */
GPULabelUpdateResult runBoundaryLPOnGPU_Chunked(
    const BoundarySubgraph& subgraph,
    const std::vector<double>& penalty,
    int num_partitions,
    size_t available_memory)
{
    GPULabelUpdateResult total_result;
    // 이터레이션 내에서 청크 간 업데이트를 반영하기 위해 라벨 배열을 복사하여 사용
    std::vector<int> new_labels = subgraph.labels;

    // 청크당 최대 노드 수를 대략적으로 계산
    double avg_degree = (subgraph.num_nodes > 0) ? (double)subgraph.num_edges / subgraph.num_nodes : 0.0;
    size_t per_node_memory = (sizeof(int) * 5) + (sizeof(int) * avg_degree);
    int max_nodes_per_chunk = std::max(1024, (int)(available_memory / (per_node_memory + 1)));

    printf("[INFO] Subgraph is too large. Starting chunked processing (%d nodes per chunk)...\n", max_nodes_per_chunk);

    for (int start_node = 0; start_node < subgraph.num_nodes; start_node += max_nodes_per_chunk) {
        int end_node = std::min(start_node + max_nodes_per_chunk, subgraph.num_nodes);
        
        BoundarySubgraph chunk_subgraph = createChunkSubgraph(subgraph, start_node, end_node);
        if (chunk_subgraph.num_nodes == 0 || chunk_subgraph.boundary_indices.empty()) continue;
        
        // mpi_rank를 전달해야 한다면, 이 함수의 인자로 추가하고 여기에도 넘겨주어야 합니다.
        GPULabelUpdateResult chunk_result = runBoundaryLPOnGPU_SubgraphUnified(
            chunk_subgraph, penalty, num_partitions);
        
        // ==================== 결과 병합 로직 (수정됨) ====================
        // chunk_result.updated_nodes 에는 '원본 그래프 ID'가 들어있습니다.
        for (size_t i = 0; i < chunk_result.updated_nodes.size(); ++i) {
            int original_node_id = chunk_result.updated_nodes[i];
            int new_label = chunk_result.updated_labels[i];

            // 원본 그래프 ID를 -> 전체 서브그래프의 인덱스로 변환합니다.
            if(original_node_id < (int)subgraph.reverse_mapping.size()) {
                int subgraph_idx = subgraph.reverse_mapping[original_node_id];

                // 변환된 인덱스가 유효하고, 현재 처리중인 청크 범위 내에 있다면 라벨을 업데이트합니다.
                if (subgraph_idx != -1 && subgraph_idx >= start_node && subgraph_idx < end_node) {
                    new_labels[subgraph_idx] = new_label;
                }
            }
        }
        // =================================================================
    }

    // 모든 청크 처리 후, 최종적으로 변경된 로컬 노드만 수집
    for(int i = 0; i < subgraph.num_nodes; ++i) {
        // local_node_flags와 node_mapping은 전체 서브그래프의 것을 사용
        if(subgraph.labels[i] != new_labels[i] && subgraph.local_node_flags[i] == 1) {
            total_result.updated_nodes.push_back(subgraph.node_mapping[i]);
            total_result.updated_labels.push_back(new_labels[i]);
            total_result.change_count++;
        }
    }
    
    return total_result;
}

/**
 * 청크용 서브그래프 생성 (수정된 최종 버전)
 */
BoundarySubgraph createChunkSubgraph(const BoundarySubgraph& original, int start_node, int end_node) {
    BoundarySubgraph chunk;
    int num_chunk_nodes = end_node - start_node;
    if (num_chunk_nodes <= 0) return chunk;

    // 1. 청크에 포함될 기본 정보 복사
    chunk.num_nodes = num_chunk_nodes;
    chunk.labels.assign(original.labels.begin() + start_node, original.labels.begin() + end_node);
    chunk.local_node_flags.assign(original.local_node_flags.begin() + start_node, original.local_node_flags.begin() + end_node);
    chunk.node_mapping.assign(original.node_mapping.begin() + start_node, original.node_mapping.begin() + end_node);

    // 2. 청크 내에서 업데이트 대상이 될 바운더리 노드들의 상대 인덱스를 찾음
    for (int boundary_original_idx : original.boundary_indices) {
        if (boundary_original_idx >= start_node && boundary_original_idx < end_node) {
            chunk.boundary_indices.push_back(boundary_original_idx - start_node);
        }
    }

    // 3. 청크의 CSR 구조를 올바르게 생성
    chunk.row_ptr.resize(chunk.num_nodes + 1, 0);
    std::vector<int> temp_col_idx;

    for (int i = 0; i < chunk.num_nodes; ++i) {
        int original_node_idx = start_node + i;
        chunk.row_ptr[i] = temp_col_idx.size();

        int start_edge = original.row_ptr[original_node_idx];
        int end_edge = original.row_ptr[original_node_idx + 1];

        for (int j = start_edge; j < end_edge; ++j) {
            int neighbor_original_idx = original.col_idx[j];
            // 중요: 이웃 노드가 현재 청크 범위 내에 있을 때만 간선을 추가
            if (neighbor_original_idx >= start_node && neighbor_original_idx < end_node) {
                temp_col_idx.push_back(neighbor_original_idx - start_node); // 청크의 상대 인덱스로 변환
            }
        }
    }
    chunk.row_ptr[chunk.num_nodes] = temp_col_idx.size();
    chunk.col_idx = std::move(temp_col_idx);
    chunk.num_edges = chunk.col_idx.size();
    
    return chunk;
}
