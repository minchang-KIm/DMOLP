#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#ifdef function
#undef function
#endif
#include <vector>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <utility>

#include "phase2/gpu_lp_boundary.h"

// ==================== 메모리 관리 및 핀드 메모리 풀 ====================
class GPUMemoryManager {
private:
    static std::vector<std::pair<void*, size_t>> allocations;
    static size_t total_allocated;
    static bool leak_detection_enabled;

public:
    static void enableLeakDetection() { leak_detection_enabled = true; }
    static void disableLeakDetection() { leak_detection_enabled = false; }
    
    static hipError_t safeMalloc(void** ptr, size_t size) {
        hipError_t err = hipMalloc(ptr, size);
        if (err == hipSuccess && leak_detection_enabled) {
            allocations.emplace_back(*ptr, size);
            total_allocated += size;
            printf("[GPU-Memory] Allocated %zu bytes at %p (total: %zu bytes)\n", 
                   size, *ptr, total_allocated);
        }
        return err;
    }
    
    static hipError_t safeFree(void* ptr) {
        if (leak_detection_enabled) {
            for (auto it = allocations.begin(); it != allocations.end(); ++it) {
                if (it->first == ptr) {
                    size_t size = it->second;
                    total_allocated -= size;
                    allocations.erase(it);
                    printf("[GPU-Memory] Freed %zu bytes at %p (total: %zu bytes)\n", 
                           size, ptr, total_allocated);
                    break;
                }
            }
        }
        return hipFree(ptr);
    }
    
    static void reportLeaks() {
        if (leak_detection_enabled && !allocations.empty()) {
            printf("[GPU-Memory-LEAK] Found %zu unfreed allocations:\n", allocations.size());
            for (const auto& p : allocations) {
                printf("  - %p: %zu bytes\n", p.first, p.second);
            }
            printf("[GPU-Memory-LEAK] Total leaked: %zu bytes\n", total_allocated);
        } else if (leak_detection_enabled) {
            printf("[GPU-Memory] No memory leaks detected!\n");
        }
    }
    
    static size_t getTotalAllocated() { return total_allocated; }
};

std::vector<std::pair<void*, size_t>> GPUMemoryManager::allocations;
size_t GPUMemoryManager::total_allocated = 0;
bool GPUMemoryManager::leak_detection_enabled = false;

class PinnedMemoryPool {
private:
    struct PinnedBuffer {
        void* host_ptr;
        void* device_ptr;
        size_t size;
        bool in_use;
        
        PinnedBuffer(size_t sz) : host_ptr(nullptr), device_ptr(nullptr), size(sz), in_use(false) {
            hipHostMalloc(&host_ptr, size);
            GPUMemoryManager::safeMalloc(&device_ptr, size);
        }
        
        ~PinnedBuffer() {
            if (host_ptr) hipHostFree(host_ptr);
            if (device_ptr) GPUMemoryManager::safeFree(device_ptr);
        }
    };
    
    static std::vector<PinnedBuffer*> buffer_pool;
    static const size_t MAX_POOL_SIZE = 10;

public:
    static PinnedBuffer* acquireBuffer(size_t size) {
        for (auto* buffer : buffer_pool) {
            if (!buffer->in_use && buffer->size >= size) {
                buffer->in_use = true;
                printf("[Pinned-Pool] Reusing buffer %p (size: %zu)\n", 
                       buffer->host_ptr, buffer->size);
                return buffer;
            }
        }
        if (buffer_pool.size() < MAX_POOL_SIZE) {
            PinnedBuffer* new_buffer = new PinnedBuffer(size);
            printf("[Pinned-Pool] Created new buffer %p (size: %zu)\n", 
                   new_buffer->host_ptr, size);
            new_buffer->in_use = true;
            buffer_pool.push_back(new_buffer);
            return new_buffer;
        }
        printf("[Pinned-Pool] Pool full, creating temporary buffer (size: %zu)\n", size);
        return new PinnedBuffer(size);
    }
    
    static void releaseBuffer(PinnedBuffer* buffer) {
        bool found = false;
        for (auto* pooled_buffer : buffer_pool) {
            if (pooled_buffer == buffer) {
                pooled_buffer->in_use = false;
                found = true;
                printf("[Pinned-Pool] Released buffer %p back to pool\n", buffer->host_ptr);
                break;
            }
        }
        if (!found) {
            printf("[Pinned-Pool] Deleting temporary buffer %p\n", buffer->host_ptr);
            delete buffer;
        }
    }
    
    static void clearPool() {
        printf("[Pinned-Pool] Clearing pool (%zu buffers)\n", buffer_pool.size());
        for (auto* buf : buffer_pool) {
            delete buf;
        }
        buffer_pool.clear();
    }
    
    static size_t getPoolSize() { return buffer_pool.size(); }
};

std::vector<PinnedMemoryPool::PinnedBuffer*> PinnedMemoryPool::buffer_pool;

// GPU용 Partition Info 구조체
struct PartitionInfoGPU {
    double P_L;   // DMOLP Penalty
};

// ==================== Warp Reduction Helper ====================
__inline__ __device__ double warpReduceSum(double val) {
    for (int offset = warpSize / 2; offset > 0; offset >>= 1)
        val += __shfl_down_sync(0xffffffff, val, offset);
    return val;
}

__inline__ __device__ int warpReduceMax(int val) {
    for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
        int other = __shfl_down_sync(0xffffffff, val, offset);
        val = max(val, other);
    }
    return val;
}

// ==================== CUDA 커널 (고성능 워프 최적화 - 기본) ====================
/**
 * 고성능 경계 노드 라벨 전파 GPU 커널
 * - 워프 협력적 처리
 * - 벡터화된 메모리 접근
 * - 공유 메모리 뱅크 충돌 최소화
 */
__global__ void boundaryLPKernel_optimized(
    const int* __restrict__ row_ptr, 
    const int* __restrict__ col_idx,
    const int* __restrict__ labels_old, 
    int* __restrict__ labels_new,
    const double* __restrict__ penalty,
    const int* __restrict__ boundary_nodes, 
    int boundary_count,
    int num_partitions,
    int labels_count)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= boundary_count) return;

    int node = boundary_nodes[idx];
    if (node < 0 || node >= labels_count) return;
    int my_label = labels_old[node];

    // 패딩된 공유 메모리 (뱅크 충돌 방지)
    extern __shared__ double shared_mem[];
    double* scores = shared_mem;
    
    // 워프 협력적 초기화
    for (int l = threadIdx.x; l < num_partitions; l += blockDim.x) {
        scores[l] = 0.0;
    }
    __syncthreads();

    // 로컬 카운터 배열 (레지스터 사용)
    double local_counts[32] = {0.0}; // 최대 32개 파티션 지원
    int max_partitions = (num_partitions < 32 ? num_partitions : 32);
    
    // 이웃 노드 순회 및 카운팅
    int start = row_ptr[node];
    int end = row_ptr[node + 1];
    
    for (int e = start; e < end; e++) {
        int neighbor = col_idx[e];
        if (neighbor >= 0 && neighbor < labels_count) {
            int neighbor_label = labels_old[neighbor];
            if (neighbor_label >= 0 && neighbor_label < max_partitions) {
                local_counts[neighbor_label] += 1.0;
            }
        }
    }
    
    // 로컬 카운트를 공유 메모리에 합산 (atomic 최소화)
    for (int l = 0; l < max_partitions; l++) {
        if (local_counts[l] > 0.0) {
            atomicAdd(&scores[l], local_counts[l]);
        }
    }
    __syncthreads();
    
    // 패널티 적용 (워프 협력)
    for (int l = threadIdx.x; l < num_partitions; l += blockDim.x) {
        if (scores[l] > 0.0) {
            scores[l] = scores[l] * (1.0 + penalty[l]);
        }
    }
    __syncthreads();

    // 워프 수준 최대값 찾기
    int best_label = my_label;
    double best_score = (my_label < num_partitions) ? scores[my_label] : 0.0;
    
    // 병렬 스캔으로 최대값 찾기
    for (int l = 0; l < num_partitions; l++) {
        if (scores[l] > best_score) {
            best_score = scores[l];
            best_label = l;
        }
    }
    
    // 결과 저장
    labels_new[node] = best_label;
}

// ==================== CUDA 커널 (안전 버전) ====================
__global__ void boundaryLPKernel_safe(
    const int* __restrict__ row_ptr, 
    const int* __restrict__ col_idx,
    const int* __restrict__ labels_old, 
    int* __restrict__ labels_new,
    const double* __restrict__ penalty,
    const int* __restrict__ boundary_nodes, 
    int boundary_count,
    int num_partitions,
    int labels_count)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= boundary_count) return;

    int node = boundary_nodes[idx];
    if (node < 0 || node >= labels_count) return;
    int my_label = labels_old[node];

    extern __shared__ double scores[];
    for (int l = threadIdx.x; l < num_partitions; l += blockDim.x) {
        scores[l] = 0.0;
    }
    __syncthreads();

    int start = row_ptr[node];
    int end = row_ptr[node + 1];
    for (int e = start; e < end; e++) {
        int neighbor = col_idx[e];
        if (neighbor >= 0 && neighbor < labels_count) {
            int neighbor_label = labels_old[neighbor];
            if (neighbor_label >= 0 && neighbor_label < num_partitions) {
                atomicAdd(&scores[neighbor_label], 1.0);
            }
        }
    }
    __syncthreads();

    for (int l = threadIdx.x; l < num_partitions; l += blockDim.x) {
        if (scores[l] > 0.0) {
            scores[l] = scores[l] * (1.0 + penalty[l]);
        }
    }
    __syncthreads();

    int best_label = my_label;
    double best_score = (my_label >= 0 && my_label < num_partitions) ? scores[my_label] : 0.0;
    for (int l = 0; l < num_partitions; l++) {
        if (scores[l] > best_score) {
            best_score = scores[l];
            best_label = l;
        }
    }
    if (best_label != my_label) {
        labels_new[node] = best_label;
    }
}

// ==================== CUDA 커널 (워프당 1-노드, 핀드 최적화 전용) ====================
__global__ void boundaryLPKernel_memory_optimized_warp(
    const int* __restrict__ row_ptr,
    const int* __restrict__ col_idx,
    const int* __restrict__ labels_old,
    int* __restrict__ labels_new,
    const double* __restrict__ penalty,
    const int* __restrict__ boundary_nodes,
    int boundary_count,
    int num_partitions,
    int labels_count)
{
    const int lane = threadIdx.x & 31;
    const int warp_id_in_block = threadIdx.x >> 5;
    const int warps_per_block = blockDim.x >> 5;

    const int node_idx = blockIdx.x * warps_per_block + warp_id_in_block;
    if (node_idx >= boundary_count) return;

    const int node = boundary_nodes[node_idx];
    if (node < 0 || node >= labels_count) return;
    const int my_label = labels_old[node];

    extern __shared__ double shm[];
    double* scores = shm + warp_id_in_block * num_partitions;

    for (int l = lane; l < num_partitions; l += 32) {
        scores[l] = 0.0;
    }
    __syncwarp();

    const int start = row_ptr[node];
    const int end   = row_ptr[node + 1];
    for (int e = start + lane; e < end; e += 32) {
        const int nei = col_idx[e];
        if (nei >= 0 && nei < labels_count) {
            const int lbl = labels_old[nei];
            if (lbl >= 0 && lbl < num_partitions) {
                atomicAdd(&scores[lbl], 1.0);
            }
        }
    }
    __syncwarp();

    for (int l = lane; l < num_partitions; l += 32) {
        if (scores[l] > 0.0) {
            scores[l] = scores[l] * (1.0 + penalty[l]);
        }
    }
    __syncwarp();

    if (lane == 0) {
        int best_label = my_label;
        double best_score = (my_label >= 0 && my_label < num_partitions) ? scores[my_label] : 0.0;
        for (int l = 0; l < num_partitions; l++) {
            if (scores[l] > best_score) {
                best_score = scores[l];
                best_label = l;
            }
        }
        if (best_label != my_label) {
            labels_new[node] = best_label;
        }
    }
}

// ==================== Public API ====================
/**
 * 고성능 GPU 라벨 전파 함수
 * 최적화된 워프 레벨 병렬성과 메모리 접근 패턴 사용
 */
void runBoundaryLPOnGPU_Optimized(
    const std::vector<int>& row_ptr,
    const std::vector<int>& col_idx,
    const std::vector<int>& labels_old,
    std::vector<int>& labels_new,
    const std::vector<double>& penalty,
    const std::vector<int>& boundary_nodes,
    int num_partitions)
{
    // 성능 측정 (CUDA 이벤트)
    hipEvent_t ev_start, ev_stop; float ms=0.0f; hipEventCreate(&ev_start); hipEventCreate(&ev_stop);
    hipEventRecord(ev_start);
    
    // CUDA 스트림 생성
    hipStream_t stream;
    hipStreamCreate(&stream);
    
    // GPU 메모리 할당
    int* d_row_ptr; int* d_col_idx;
    int* d_labels_old; int* d_labels_new;
    int* d_boundary;
    double* d_penalty;

    size_t row_size = row_ptr.size() * sizeof(int);
    size_t col_size = col_idx.size() * sizeof(int);
    size_t labels_size = labels_old.size() * sizeof(int);
    size_t boundary_size = boundary_nodes.size() * sizeof(int);
    size_t penalty_size = penalty.size() * sizeof(double);

    hipMalloc(&d_row_ptr, row_size);
    hipMalloc(&d_col_idx, col_size);
    hipMalloc(&d_labels_old, labels_size);
    hipMalloc(&d_labels_new, labels_size);
    hipMalloc(&d_boundary, boundary_size);
    hipMalloc(&d_penalty, penalty_size);

    // 비동기 메모리 복사
    hipMemcpyAsync(d_row_ptr, row_ptr.data(), row_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_col_idx, col_idx.data(), col_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_labels_old, labels_old.data(), labels_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_labels_new, labels_new.data(), labels_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_boundary, boundary_nodes.data(), boundary_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_penalty, penalty.data(), penalty_size, hipMemcpyHostToDevice, stream);

    // 최적화된 커널 실행 설정
    int threads = 256;  // 높은 occupancy
    int blocks = (boundary_nodes.size() + threads - 1) / threads;
    size_t shared_mem = (num_partitions + 32) * sizeof(double);  // 뱅크 충돌 방지

    // 커널 실행
    boundaryLPKernel_optimized<<<blocks, threads, shared_mem, stream>>>(
        d_row_ptr, d_col_idx,
        d_labels_old, d_labels_new,
        d_penalty,
        d_boundary, boundary_nodes.size(),
        num_partitions,
        static_cast<int>(labels_old.size()));

    // 결과 복사
    hipMemcpyAsync(labels_new.data(), d_labels_new, labels_size, 
                    hipMemcpyDeviceToHost, stream);
    
    // 동기화 및 시간 측정 종료
    hipStreamSynchronize(stream);
    hipEventRecord(ev_stop); hipEventSynchronize(ev_stop); hipEventElapsedTime(&ms, ev_start, ev_stop);
    
    printf("[GPU-Optimized] Execution time: %ld μs (boundary nodes: %zu)\n", 
           (long)(ms*1000.0f), boundary_nodes.size());
    hipEventDestroy(ev_start); hipEventDestroy(ev_stop);

    // 정리
    hipStreamDestroy(stream);
    hipFree(d_row_ptr);
    hipFree(d_col_idx);
    hipFree(d_labels_old);
    hipFree(d_labels_new);
    hipFree(d_boundary);
    hipFree(d_penalty);
}

// 안전한 GPU 라벨 전파 (fallback)
void runBoundaryLPOnGPU_Safe(
    const std::vector<int>& row_ptr,
    const std::vector<int>& col_idx,
    const std::vector<int>& labels_old,
    std::vector<int>& labels_new,
    const std::vector<double>& penalty,
    const std::vector<int>& boundary_nodes,
    int num_partitions)
{
    hipEvent_t ev_start, ev_stop; float ms=0.0f; hipEventCreate(&ev_start); hipEventCreate(&ev_stop);
    hipEventRecord(ev_start);
    hipStream_t stream; hipStreamCreate(&stream);

    int *d_row_ptr, *d_col_idx, *d_labels_old, *d_labels_new, *d_boundary; double* d_penalty;
    size_t row_size = row_ptr.size() * sizeof(int);
    size_t col_size = col_idx.size() * sizeof(int);
    size_t labels_size = labels_old.size() * sizeof(int);
    size_t boundary_size = boundary_nodes.size() * sizeof(int);
    size_t penalty_size = penalty.size() * sizeof(double);

    hipMalloc(&d_row_ptr, row_size);
    hipMalloc(&d_col_idx, col_size);
    hipMalloc(&d_labels_old, labels_size);
    hipMalloc(&d_labels_new, labels_size);
    hipMalloc(&d_boundary, boundary_size);
    hipMalloc(&d_penalty, penalty_size);

    hipMemcpyAsync(d_row_ptr, row_ptr.data(), row_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_col_idx, col_idx.data(), col_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_labels_old, labels_old.data(), labels_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_labels_new, labels_new.data(), labels_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_boundary, boundary_nodes.data(), boundary_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_penalty, penalty.data(), penalty_size, hipMemcpyHostToDevice, stream);

    int threads = 256;
    int blocks = (boundary_nodes.size() + threads - 1) / threads;
    size_t shared_mem = (num_partitions + 8) * sizeof(double);

    boundaryLPKernel_safe<<<blocks, threads, shared_mem, stream>>>(
        d_row_ptr, d_col_idx, d_labels_old, d_labels_new, d_penalty, d_boundary,
        boundary_nodes.size(), num_partitions,
        static_cast<int>(labels_old.size()));

    hipMemcpyAsync(labels_new.data(), d_labels_new, labels_size, hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    hipEventRecord(ev_stop); hipEventSynchronize(ev_stop); hipEventElapsedTime(&ms, ev_start, ev_stop);
    printf("[GPU-Safe] Execution time: %ld μs (boundary nodes: %zu)\n", (long)(ms*1000.0f), boundary_nodes.size());
    hipEventDestroy(ev_start); hipEventDestroy(ev_stop);

    hipStreamDestroy(stream);
    hipFree(d_row_ptr); hipFree(d_col_idx); hipFree(d_labels_old);
    hipFree(d_labels_new); hipFree(d_boundary); hipFree(d_penalty);
}

// 핀드 메모리 최적화 경계 LP (워프당 1-노드)
void runBoundaryLPOnGPU_PinnedOptimized(
    const std::vector<int>& row_ptr,
    const std::vector<int>& col_idx,
    const std::vector<int>& labels_old,
    std::vector<int>& labels_new,
    const std::vector<double>& penalty,
    const std::vector<int>& boundary_nodes,
    int num_partitions)
{
    GPUMemoryManager::enableLeakDetection();
    hipEvent_t ev_start, ev_stop; float ms=0.0f; hipEventCreate(&ev_start); hipEventCreate(&ev_stop);
    hipEventRecord(ev_start);
    
    try {
        hipStream_t stream; hipStreamCreate(&stream);
        
        size_t row_size = row_ptr.size() * sizeof(int);
        size_t col_size = col_idx.size() * sizeof(int);
        size_t labels_size = labels_old.size() * sizeof(int);
        size_t boundary_size = boundary_nodes.size() * sizeof(int);
        size_t penalty_size = penalty.size() * sizeof(double);
        
        auto row_buffer = PinnedMemoryPool::acquireBuffer(row_size);
        auto col_buffer = PinnedMemoryPool::acquireBuffer(col_size);
        auto labels_old_buffer = PinnedMemoryPool::acquireBuffer(labels_size);
        auto labels_new_buffer = PinnedMemoryPool::acquireBuffer(labels_size);
        auto boundary_buffer = PinnedMemoryPool::acquireBuffer(boundary_size);
        auto penalty_buffer = PinnedMemoryPool::acquireBuffer(penalty_size);
        
        memcpy(row_buffer->host_ptr, row_ptr.data(), row_size);
        memcpy(col_buffer->host_ptr, col_idx.data(), col_size);
        memcpy(labels_old_buffer->host_ptr, labels_old.data(), labels_size);
        memcpy(labels_new_buffer->host_ptr, labels_new.data(), labels_size);
        memcpy(boundary_buffer->host_ptr, boundary_nodes.data(), boundary_size);
        memcpy(penalty_buffer->host_ptr, penalty.data(), penalty_size);
        
        hipMemcpyAsync(row_buffer->device_ptr, row_buffer->host_ptr, row_size, hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(col_buffer->device_ptr, col_buffer->host_ptr, col_size, hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(labels_old_buffer->device_ptr, labels_old_buffer->host_ptr, labels_size, hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(labels_new_buffer->device_ptr, labels_new_buffer->host_ptr, labels_size, hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(boundary_buffer->device_ptr, boundary_buffer->host_ptr, boundary_size, hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(penalty_buffer->device_ptr, penalty_buffer->host_ptr, penalty_size, hipMemcpyHostToDevice, stream);
        
        int warpsPerBlock = 8;
        int blockSize = warpsPerBlock * 32;
        int gridSize = (static_cast<int>(boundary_nodes.size()) + warpsPerBlock - 1) / warpsPerBlock;
        size_t shared_mem = static_cast<size_t>(num_partitions) * warpsPerBlock * sizeof(double);

        boundaryLPKernel_memory_optimized_warp<<<gridSize, blockSize, shared_mem, stream>>>(
            (int*)row_buffer->device_ptr, (int*)col_buffer->device_ptr,
            (int*)labels_old_buffer->device_ptr, (int*)labels_new_buffer->device_ptr,
            (double*)penalty_buffer->device_ptr,
            (int*)boundary_buffer->device_ptr, boundary_nodes.size(),
            num_partitions,
            static_cast<int>(labels_old.size()));
        
        hipMemcpyAsync(labels_new_buffer->host_ptr, labels_new_buffer->device_ptr, labels_size, hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
        memcpy(labels_new.data(), labels_new_buffer->host_ptr, labels_size);
        
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("[GPU-Pinned-Error] CUDA error: %s\n", hipGetErrorString(error));
        }
        
        hipStreamDestroy(stream);
        PinnedMemoryPool::releaseBuffer(row_buffer);
        PinnedMemoryPool::releaseBuffer(col_buffer);
        PinnedMemoryPool::releaseBuffer(labels_old_buffer);
        PinnedMemoryPool::releaseBuffer(labels_new_buffer);
        PinnedMemoryPool::releaseBuffer(boundary_buffer);
        PinnedMemoryPool::releaseBuffer(penalty_buffer);
        
    } catch (const std::exception& e) {
        printf("[GPU-Pinned-Exception] %s\n", e.what());
    }
    
    hipEventRecord(ev_stop); hipEventSynchronize(ev_stop); hipEventElapsedTime(&ms, ev_start, ev_stop);
    printf("[GPU-Pinned-Optimized] Execution time: %ld μs (boundary nodes: %zu)\n", (long)(ms*1000.0f), boundary_nodes.size());
    hipEventDestroy(ev_start); hipEventDestroy(ev_stop);
    printf("[GPU-Pinned-Pool] Pool size: %zu buffers\n", PinnedMemoryPool::getPoolSize());
    
}

// 리소스 정리
void cleanupGPUResources() {
    printf("[GPU-Cleanup] Cleaning up all GPU resources...\n");
    PinnedMemoryPool::clearPool();
    GPUMemoryManager::reportLeaks();
    printf("[GPU-Cleanup] Cleanup complete.\n");
}