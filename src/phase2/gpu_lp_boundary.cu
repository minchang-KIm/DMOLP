#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#ifdef function
#undef function
#endif
#include <vector>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <utility>
#include <unordered_set>
#include <unordered_map>
#include <algorithm>
#include <cstring>
#include <omp.h>

#include "phase2/gpu_lp_boundary.h"

// ==================== 바운더리 서브그래프 생성 (로컬+고스트 통합) ====================

/**
 * 바운더리 노드 + 1-hop 이웃으로 구성된 통합 서브그래프 생성
 * - 로컬 노드와 고스트 노드의 라벨을 통합하여 GPU에 전달
 * - 로컬 노드만 업데이트 대상으로 표시
 * - GPU 메모리 지역성 최적화
 */
BoundarySubgraph createBoundarySubgraphUnified(
    const std::vector<int>& row_ptr,
    const std::vector<int>& col_idx,
    const std::vector<int>& boundary_nodes,
    const std::vector<int>& local_labels,
    const std::vector<int>& ghost_labels,
    const std::vector<int>& global_ids,
    int num_local_nodes)
{
    BoundarySubgraph subgraph;
    subgraph.num_local_nodes = num_local_nodes;
    
    // 1단계: 서브그래프에 포함될 모든 노드 수집 (바운더리 + 1-hop 이웃)
    std::unordered_set<int> subgraph_nodes_set;
    
    // 바운더리 노드들 추가
    for (int boundary_node : boundary_nodes) {
        if (boundary_node >= 0 && boundary_node < (int)(local_labels.size() + ghost_labels.size())) {
            subgraph_nodes_set.insert(boundary_node);
        }
    }
    
    // 각 바운더리 노드의 1-hop 이웃 추가
    for (int boundary_node : boundary_nodes) {
        if (boundary_node >= 0 && boundary_node < (int)row_ptr.size() - 1) {
            for (int edge_idx = row_ptr[boundary_node]; edge_idx < row_ptr[boundary_node + 1]; edge_idx++) {
                int neighbor = col_idx[edge_idx];
                if (neighbor >= 0 && neighbor < (int)(local_labels.size() + ghost_labels.size())) {
                    subgraph_nodes_set.insert(neighbor);
                }
            }
        }
    }
    
    // 2단계: 노드 매핑 구성
    std::vector<int> subgraph_nodes(subgraph_nodes_set.begin(), subgraph_nodes_set.end());
    std::sort(subgraph_nodes.begin(), subgraph_nodes.end());
    
    subgraph.num_nodes = subgraph_nodes.size();
    subgraph.node_mapping = subgraph_nodes;
    subgraph.reverse_mapping.resize(local_labels.size() + ghost_labels.size(), -1);
    
    // 통합 라벨 배열 및 로컬 노드 플래그 구성
    subgraph.labels.resize(subgraph.num_nodes);
    subgraph.local_node_flags.resize(subgraph.num_nodes);
    
    for (int i = 0; i < subgraph.num_nodes; i++) {
        int orig_node = subgraph_nodes[i];
        subgraph.reverse_mapping[orig_node] = i;
        
        // 라벨 설정 (로컬 또는 고스트)
        if (orig_node < num_local_nodes) {
            // 로컬 노드
            subgraph.labels[i] = local_labels[orig_node];
            subgraph.local_node_flags[i] = 1;
        } else {
            // 고스트 노드
            int ghost_idx = orig_node - num_local_nodes;
            if (ghost_idx >= 0 && ghost_idx < (int)ghost_labels.size()) {
                subgraph.labels[i] = ghost_labels[ghost_idx];
            } else {
                subgraph.labels[i] = -1; // 유효하지 않은 라벨
            }
            subgraph.local_node_flags[i] = 0;
        }
    }
    
    // 3단계: 서브그래프 CSR 구성
    subgraph.row_ptr.resize(subgraph.num_nodes + 1, 0);
    std::vector<std::vector<int>> adj_list(subgraph.num_nodes);
    
    // 각 서브그래프 노드에 대해 이웃 수집
    for (int i = 0; i < subgraph.num_nodes; i++) {
        int orig_node = subgraph_nodes[i];
        if (orig_node < (int)row_ptr.size() - 1) {
            for (int edge_idx = row_ptr[orig_node]; edge_idx < row_ptr[orig_node + 1]; edge_idx++) {
                int neighbor = col_idx[edge_idx];
                if (neighbor >= 0 && neighbor < (int)(local_labels.size() + ghost_labels.size())) {
                    int neighbor_subgraph_idx = subgraph.reverse_mapping[neighbor];
                    if (neighbor_subgraph_idx != -1) {
                        adj_list[i].push_back(neighbor_subgraph_idx);
                    }
                }
            }
        }
    }
    
    // CSR 형태로 변환
    int edge_count = 0;
    for (int i = 0; i < subgraph.num_nodes; i++) {
        subgraph.row_ptr[i] = edge_count;
        edge_count += adj_list[i].size();
    }
    subgraph.row_ptr[subgraph.num_nodes] = edge_count;
    subgraph.num_edges = edge_count;
    
    subgraph.col_idx.resize(edge_count);
    int idx = 0;
    for (int i = 0; i < subgraph.num_nodes; i++) {
        for (int neighbor : adj_list[i]) {
            subgraph.col_idx[idx++] = neighbor;
        }
    }
    
    // 4단계: 서브그래프 내 실제 바운더리 노드 인덱스 찾기 (로컬 노드만)
    std::unordered_set<int> boundary_set(boundary_nodes.begin(), boundary_nodes.end());
    for (int i = 0; i < subgraph.num_nodes; i++) {
        int orig_node = subgraph_nodes[i];
        if (boundary_set.count(orig_node) && subgraph.local_node_flags[i] == 1) {
            subgraph.boundary_indices.push_back(i);
        }
    }
    
    int local_count = 0;
    for (int flag : subgraph.local_node_flags) {
        if (flag == 1) local_count++;
    }
    
    printf("[Subgraph-Unified] Created: %d nodes (%d local, %d ghost), %d edges, %zu boundary nodes\n", 
           subgraph.num_nodes, local_count, subgraph.num_nodes - local_count, 
           subgraph.num_edges, subgraph.boundary_indices.size());
    
    return subgraph;
}

/**
 * 적응적 바운더리 확장: 이전 바운더리 + 1-hop 이웃에서 실제 바운더리만 필터링
 */
std::vector<int> expandBoundaryNodes(
    const std::vector<int>& row_ptr,
    const std::vector<int>& col_idx,
    const std::vector<int>& prev_boundary_nodes,
    const std::vector<int>& labels,
    int labels_count)
{
    std::unordered_set<int> candidate_nodes;
    
    // 이전 바운더리 노드들과 그들의 1-hop 이웃 수집
    for (int boundary_node : prev_boundary_nodes) {
        if (boundary_node >= 0 && boundary_node < labels_count) {
            candidate_nodes.insert(boundary_node);
            
            if (boundary_node < (int)row_ptr.size() - 1) {
                for (int edge_idx = row_ptr[boundary_node]; edge_idx < row_ptr[boundary_node + 1]; edge_idx++) {
                    int neighbor = col_idx[edge_idx];
                    if (neighbor >= 0 && neighbor < labels_count) {
                        candidate_nodes.insert(neighbor);
                    }
                }
            }
        }
    }
    
    // 후보 노드들 중에서 실제 바운더리 노드만 필터링
    std::vector<int> new_boundary_nodes;
    for (int node : candidate_nodes) {
        if (node >= 0 && node < labels_count && node < (int)row_ptr.size() - 1) {
            int node_label = labels[node];
            bool is_boundary = false;
            
            for (int edge_idx = row_ptr[node]; edge_idx < row_ptr[node + 1]; edge_idx++) {
                int neighbor = col_idx[edge_idx];
                if (neighbor >= 0 && neighbor < labels_count) {
                    int neighbor_label = labels[neighbor];
                    if (neighbor_label != node_label) {
                        is_boundary = true;
                        break;
                    }
                }
            }
            
            if (is_boundary) {
                new_boundary_nodes.push_back(node);
            }
        }
    }
    
    printf("[Boundary-Expansion] %zu -> %zu nodes\n", 
           prev_boundary_nodes.size(), new_boundary_nodes.size());
    
    return new_boundary_nodes;
}

// ==================== 메모리 관리 및 핀드 메모리 풀 ====================
class GPUMemoryManager {
private:
    static std::vector<std::pair<void*, size_t>> allocations;
    static size_t total_allocated;
    static bool leak_detection_enabled;

public:
    static void enableLeakDetection() { leak_detection_enabled = true; }
    static void disableLeakDetection() { leak_detection_enabled = false; }
    
    static hipError_t safeMalloc(void** ptr, size_t size) {
        hipError_t err = hipMalloc(ptr, size);
        if (err == hipSuccess && leak_detection_enabled) {
            allocations.emplace_back(*ptr, size);
            total_allocated += size;
            printf("[GPU-Memory] Allocated %zu bytes at %p (total: %zu bytes)\n", 
                   size, *ptr, total_allocated);
        }
        return err;
    }
    
    static hipError_t safeFree(void* ptr) {
        if (leak_detection_enabled) {
            for (auto it = allocations.begin(); it != allocations.end(); ++it) {
                if (it->first == ptr) {
                    size_t size = it->second;
                    total_allocated -= size;
                    allocations.erase(it);
                    printf("[GPU-Memory] Freed %zu bytes at %p (total: %zu bytes)\n", 
                           size, ptr, total_allocated);
                    break;
                }
            }
        }
        return hipFree(ptr);
    }
    
    static void reportLeaks() {
        if (leak_detection_enabled && !allocations.empty()) {
            printf("[GPU-Memory-LEAK] Found %zu unfreed allocations:\n", allocations.size());
            for (const auto& p : allocations) {
                printf("  - %p: %zu bytes\n", p.first, p.second);
            }
            printf("[GPU-Memory-LEAK] Total leaked: %zu bytes\n", total_allocated);
        } else if (leak_detection_enabled) {
            printf("[GPU-Memory] No memory leaks detected!\n");
        }
    }
    
    static size_t getTotalAllocated() { return total_allocated; }
};

std::vector<std::pair<void*, size_t>> GPUMemoryManager::allocations;
size_t GPUMemoryManager::total_allocated = 0;
bool GPUMemoryManager::leak_detection_enabled = false;

class PinnedMemoryPool {
private:
    struct PinnedBuffer {
        void* host_ptr;
        void* device_ptr;
        size_t size;
        bool in_use;
        
        PinnedBuffer(size_t sz) : host_ptr(nullptr), device_ptr(nullptr), size(sz), in_use(false) {
            hipHostMalloc(&host_ptr, size);
            GPUMemoryManager::safeMalloc(&device_ptr, size);
        }
        
        ~PinnedBuffer() {
            if (host_ptr) hipHostFree(host_ptr);
            if (device_ptr) GPUMemoryManager::safeFree(device_ptr);
        }
    };
    
    static std::vector<PinnedBuffer*> buffer_pool;
    static const size_t MAX_POOL_SIZE = 10;

public:
    static PinnedBuffer* acquireBuffer(size_t size) {
        for (auto* buffer : buffer_pool) {
            if (!buffer->in_use && buffer->size >= size) {
                buffer->in_use = true;
                printf("[Pinned-Pool] Reusing buffer %p (size: %zu)\n", 
                       buffer->host_ptr, buffer->size);
                return buffer;
            }
        }
        if (buffer_pool.size() < MAX_POOL_SIZE) {
            PinnedBuffer* new_buffer = new PinnedBuffer(size);
            printf("[Pinned-Pool] Created new buffer %p (size: %zu)\n", 
                   new_buffer->host_ptr, size);
            new_buffer->in_use = true;
            buffer_pool.push_back(new_buffer);
            return new_buffer;
        }
        printf("[Pinned-Pool] Pool full, creating temporary buffer (size: %zu)\n", size);
        return new PinnedBuffer(size);
    }
    
    static void releaseBuffer(PinnedBuffer* buffer) {
        bool found = false;
        for (auto* pooled_buffer : buffer_pool) {
            if (pooled_buffer == buffer) {
                pooled_buffer->in_use = false;
                found = true;
                printf("[Pinned-Pool] Released buffer %p back to pool\n", buffer->host_ptr);
                break;
            }
        }
        if (!found) {
            printf("[Pinned-Pool] Deleting temporary buffer %p\n", buffer->host_ptr);
            delete buffer;
        }
    }
    
    static void clearPool() {
        printf("[Pinned-Pool] Clearing pool (%zu buffers)\n", buffer_pool.size());
        for (auto* buf : buffer_pool) {
            delete buf;
        }
        buffer_pool.clear();
    }
    
    static size_t getPoolSize() { return buffer_pool.size(); }
};

std::vector<PinnedMemoryPool::PinnedBuffer*> PinnedMemoryPool::buffer_pool;

// ==================== 서브그래프 전용 커널 (로컬 노드만 업데이트) ====================

/**
 * 통합 서브그래프 전용 최적화 커널
 * - 로컬+고스트 라벨을 모두 참조하지만 로컬 노드만 업데이트
 * - 로컬 노드 플래그로 업데이트 대상 구분
 */
__global__ void boundaryLPKernel_unified(
    const int* __restrict__ row_ptr, 
    const int* __restrict__ col_idx,
    const int* __restrict__ labels_old, 
    int* __restrict__ labels_new,
    const int* __restrict__ local_node_flags,
    const double* __restrict__ penalty,
    const int* __restrict__ boundary_indices, 
    int boundary_count,
    int num_partitions,
    int subgraph_size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= boundary_count) return;

    int subgraph_node_idx = boundary_indices[idx];
    if (subgraph_node_idx < 0 || subgraph_node_idx >= subgraph_size) return;
    
    // 로컬 노드가 아니면 스킵 (고스트 노드는 업데이트하지 않음)
    if (local_node_flags[subgraph_node_idx] != 1) return;
    
    int my_label = labels_old[subgraph_node_idx];

    extern __shared__ double scores[];
    
    // 공유 메모리 초기화 (워프 협력)
    for (int l = threadIdx.x; l < num_partitions; l += blockDim.x) {
        scores[l] = 0.0;
    }
    __syncthreads();

    // 로컬 카운터 (레지스터 사용)
    double local_counts[32] = {0.0};
    int max_partitions = (num_partitions < 32 ? num_partitions : 32);
    
    // 이웃 노드 순회 (서브그래프 내에서 로컬+고스트 모두 참조)
    int start = row_ptr[subgraph_node_idx];
    int end = row_ptr[subgraph_node_idx + 1];
    
    for (int e = start; e < end; e++) {
        int neighbor_idx = col_idx[e];
        if (neighbor_idx >= 0 && neighbor_idx < subgraph_size) {
            int neighbor_label = labels_old[neighbor_idx];
            if (neighbor_label >= 0 && neighbor_label < max_partitions) {
                local_counts[neighbor_label] += 1.0;
            }
        }
    }
    
    // 공유 메모리에 합산
    for (int l = 0; l < max_partitions; l++) {
        if (local_counts[l] > 0.0) {
            atomicAdd(&scores[l], local_counts[l]);
        }
    }
    __syncthreads();
    
    // 패널티 적용
    for (int l = threadIdx.x; l < num_partitions; l += blockDim.x) {
        if (scores[l] > 0.0) {
            scores[l] = scores[l] * (1.0 + penalty[l]);
        }
    }
    __syncthreads();

    // 최적 라벨 찾기
    int best_label = my_label;
    double best_score = (my_label >= 0 && my_label < num_partitions) ? scores[my_label] : 0.0;
    
    for (int l = 0; l < num_partitions; l++) {
        if (scores[l] > best_score) {
            best_score = scores[l];
            best_label = l;
        }
    }
    
    // 결과 저장 (로컬 노드만)
    labels_new[subgraph_node_idx] = best_label;
}

// ==================== 효율적인 GPU 처리 함수 ====================

/**
 * 효율적인 GPU 라벨 전파 (통합 서브그래프, 로컬만 업데이트)
 * - 전체 그래프 대신 서브그래프만 GPU에 전달
 * - 로컬+고스트 라벨 통합으로 GPU 메모리 지역성 최적화
 * - 로컬 노드만 업데이트하여 정확성 보장
 */
GPULabelUpdateResult runBoundaryLPOnGPU_SubgraphUnified(
    const BoundarySubgraph& subgraph,
    const std::vector<double>& penalty,
    int num_partitions)
{
    GPULabelUpdateResult result;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    hipStream_t stream;
    hipStreamCreate(&stream);
    
    // GPU 메모리 할당 (서브그래프 크기만)
    int *d_row_ptr, *d_col_idx, *d_labels_old, *d_labels_new;
    int *d_local_flags, *d_boundary_indices;
    double *d_penalty;
    
    GPUMemoryManager::safeMalloc((void**)&d_row_ptr, subgraph.row_ptr.size() * sizeof(int));
    GPUMemoryManager::safeMalloc((void**)&d_col_idx, subgraph.col_idx.size() * sizeof(int));
    GPUMemoryManager::safeMalloc((void**)&d_labels_old, subgraph.num_nodes * sizeof(int));
    GPUMemoryManager::safeMalloc((void**)&d_labels_new, subgraph.num_nodes * sizeof(int));
    GPUMemoryManager::safeMalloc((void**)&d_local_flags, subgraph.num_nodes * sizeof(int));
    GPUMemoryManager::safeMalloc((void**)&d_boundary_indices, subgraph.boundary_indices.size() * sizeof(int));
    GPUMemoryManager::safeMalloc((void**)&d_penalty, penalty.size() * sizeof(double));
    
    // 비동기 메모리 전송
    hipMemcpyAsync(d_row_ptr, subgraph.row_ptr.data(), subgraph.row_ptr.size() * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_col_idx, subgraph.col_idx.data(), subgraph.col_idx.size() * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_labels_old, subgraph.labels.data(), subgraph.num_nodes * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_labels_new, subgraph.labels.data(), subgraph.num_nodes * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_local_flags, subgraph.local_node_flags.data(), subgraph.num_nodes * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_boundary_indices, subgraph.boundary_indices.data(), subgraph.boundary_indices.size() * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_penalty, penalty.data(), penalty.size() * sizeof(double), hipMemcpyHostToDevice, stream);
    
    // 커널 실행 설정 (GPU 하드웨어 적응형)
    int threads = 256;
    int boundary_count = subgraph.boundary_indices.size();
    
    // GPU 멀티프로세서 정보 가져오기
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    
    // 최적 블록 개수 계산 (SM 개수와 바운더리 노드 개수 고려)
    int max_blocks_per_sm = prop.maxThreadsPerMultiProcessor / threads;
    int max_blocks = prop.multiProcessorCount * max_blocks_per_sm;
    int min_blocks_needed = (boundary_count + threads - 1) / threads;
    int blocks = std::min(max_blocks, min_blocks_needed);
    
    printf("[GPU] SM개수: %d, 최대블록: %d, 필요블록: %d, 사용블록: %d (바운더리: %d)\n",
           prop.multiProcessorCount, max_blocks, min_blocks_needed, blocks, boundary_count);
    
    size_t shared_mem = num_partitions * sizeof(double);
    
    boundaryLPKernel_unified<<<blocks, threads, shared_mem, stream>>>(
        d_row_ptr, d_col_idx, d_labels_old, d_labels_new, d_local_flags, d_penalty,
        d_boundary_indices, subgraph.boundary_indices.size(),
        num_partitions, subgraph.num_nodes);
    
    // 결과 복사
    std::vector<int> updated_labels(subgraph.num_nodes);
    hipMemcpyAsync(updated_labels.data(), d_labels_new, subgraph.num_nodes * sizeof(int), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    
    // 변경된 로컬 노드만 추출
    for (int i = 0; i < subgraph.num_nodes; i++) {
        if (subgraph.local_node_flags[i] == 1) { // 로컬 노드만
            if (subgraph.labels[i] != updated_labels[i]) { // 라벨이 변경된 경우
                int orig_node_id = subgraph.node_mapping[i];
                result.updated_nodes.push_back(orig_node_id);
                result.updated_labels.push_back(updated_labels[i]);
                result.change_count++;
            }
        }
    }
    
    // 정리
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    
    printf("[GPU-Unified] Execution time: %.2f ms (%d nodes, %d boundary, %d changes)\n", 
           ms, subgraph.num_nodes, (int)subgraph.boundary_indices.size(), result.change_count);
    
    hipStreamDestroy(stream);
    GPUMemoryManager::safeFree(d_row_ptr);
    GPUMemoryManager::safeFree(d_col_idx);
    GPUMemoryManager::safeFree(d_labels_old);
    GPUMemoryManager::safeFree(d_labels_new);
    GPUMemoryManager::safeFree(d_local_flags);
    GPUMemoryManager::safeFree(d_boundary_indices);
    GPUMemoryManager::safeFree(d_penalty);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return result;
}

/**
 * 스트리밍 방식 GPU 처리 (개선된 인터페이스)
 */
GPULabelUpdateResult runBoundaryLPOnGPU_Streaming(
    const std::vector<int>& row_ptr,
    const std::vector<int>& col_idx,
    const std::vector<int>& boundary_nodes,
    const std::vector<int>& local_labels,
    const std::vector<int>& ghost_labels,
    const std::vector<int>& global_ids,
    const std::vector<double>& penalty,
    int num_local_nodes,
    int num_partitions,
    size_t max_memory_mb)
{
    printf("[GPU-Streaming] Starting with %zu boundary nodes, memory limit: %zu MB\n", 
           boundary_nodes.size(), max_memory_mb);
    
    // 통합 서브그래프 생성
    BoundarySubgraph subgraph = createBoundarySubgraphUnified(
        row_ptr, col_idx, boundary_nodes, local_labels, ghost_labels, global_ids, num_local_nodes);
    
    // 메모리 사용량 계산
    size_t subgraph_memory = (subgraph.row_ptr.size() + subgraph.col_idx.size() + 
                             subgraph.num_nodes * 4) * sizeof(int) + // CSR + labels + flags + boundary_indices 
                            penalty.size() * sizeof(double);
    
    size_t available_memory = max_memory_mb * 1024 * 1024;
    
    if (subgraph_memory <= available_memory) {
        // 전체 서브그래프가 메모리에 들어가는 경우
        printf("[GPU-Streaming] Processing entire subgraph (%zu bytes)\n", subgraph_memory);
        return runBoundaryLPOnGPU_SubgraphUnified(subgraph, penalty, num_partitions);
    } else {
        // 청크 단위 처리 필요
        printf("[GPU-Streaming] Subgraph too large (%zu bytes), using chunking\n", subgraph_memory);
        return runBoundaryLPOnGPU_Chunked(subgraph, penalty, num_partitions, available_memory);
    }
}

/**
 * 청크 단위 처리 함수 구현
 * 큰 서브그래프를 메모리에 맞게 청크로 나누어 처리
 */
GPULabelUpdateResult runBoundaryLPOnGPU_Chunked(
    const BoundarySubgraph& subgraph,
    const std::vector<double>& penalty,
    int num_partitions,
    size_t available_memory) {
    
    // 결과 누적용
    GPULabelUpdateResult total_result;
    
    // 각 청크당 최대 노드 수 계산
    size_t per_node_memory = sizeof(int) * 3 + sizeof(bool); // CSR + labels + flags
    size_t overhead_memory = penalty.size() * sizeof(double) + 1024 * 1024; // penalty + 1MB 오버헤드
    size_t usable_memory = available_memory - overhead_memory;
    int max_nodes_per_chunk = std::max(1, (int)(usable_memory / per_node_memory));
    
    printf("[GPU-Chunked] Processing %d nodes in chunks of max %d nodes\n", 
           subgraph.num_nodes, max_nodes_per_chunk);
    
    // 노드 범위별로 청크 처리
    for (int start_node = 0; start_node < subgraph.num_nodes; start_node += max_nodes_per_chunk) {
        int end_node = std::min(start_node + max_nodes_per_chunk, subgraph.num_nodes);
        int chunk_size = end_node - start_node;
        
        printf("[GPU-Chunked] Processing chunk [%d-%d] (%d nodes)\n", 
               start_node, end_node-1, chunk_size);
        
        // 청크용 서브그래프 생성
        BoundarySubgraph chunk_subgraph = createChunkSubgraph(subgraph, start_node, end_node);
        
        // 청크 처리
        GPULabelUpdateResult chunk_result = runBoundaryLPOnGPU_SubgraphUnified(
            chunk_subgraph, penalty, num_partitions);
        
        // 결과 병합 (노드 인덱스를 원래 인덱스로 변환)
        for (size_t i = 0; i < chunk_result.updated_nodes.size(); i++) {
            int original_node = chunk_result.updated_nodes[i] + start_node;
            if (original_node < subgraph.num_local_nodes) { // 로컬 노드만
                total_result.updated_nodes.push_back(original_node);
                total_result.updated_labels.push_back(chunk_result.updated_labels[i]);
            }
        }
        
        // 청크 메모리 해제
        freeChunkSubgraph(chunk_subgraph);
    }
    
    printf("[GPU-Chunked] Total updated nodes: %zu\n", total_result.updated_nodes.size());
    return total_result;
}

/**
 * 청크용 서브그래프 생성
 */
BoundarySubgraph createChunkSubgraph(const BoundarySubgraph& original, int start_node, int end_node) {
    BoundarySubgraph chunk;
    chunk.num_nodes = end_node - start_node;
    chunk.num_local_nodes = 0;
    
    // 로컬 노드 수 계산
    for (int i = start_node; i < end_node; i++) {
        if (original.local_node_flags[i]) {
            chunk.num_local_nodes++;
        }
    }
    
    // 청크의 엣지 수 계산
    chunk.num_edges = 0;
    for (int i = start_node; i < end_node; i++) {
        chunk.num_edges += original.row_ptr[i+1] - original.row_ptr[i];
    }
    
    // 메모리 할당
    chunk.row_ptr.resize(chunk.num_nodes + 1);
    chunk.col_idx.resize(chunk.num_edges);
    chunk.labels.resize(chunk.num_nodes);
    chunk.local_node_flags.resize(chunk.num_nodes);
    
    // 데이터 복사
    chunk.row_ptr[0] = 0;
    int edge_offset = 0;
    
    for (int i = 0; i < chunk.num_nodes; i++) {
        int original_node = start_node + i;
        
        // 라벨과 플래그 복사
        chunk.labels[i] = original.labels[original_node];
        chunk.local_node_flags[i] = original.local_node_flags[original_node];
        
        // 엣지 정보 복사
        int start_edge = original.row_ptr[original_node];
        int end_edge = original.row_ptr[original_node + 1];
        int edge_count = end_edge - start_edge;
        
        for (int j = 0; j < edge_count; j++) {
            chunk.col_idx[edge_offset + j] = 
                original.col_idx[start_edge + j] - start_node; // 상대 인덱스로 변환
        }
        
        edge_offset += edge_count;
        chunk.row_ptr[i + 1] = edge_offset;
    }
    
    return chunk;
}

/**
 * 청크 서브그래프 메모리 해제
 */
void freeChunkSubgraph(BoundarySubgraph& chunk) {
    // std::vector는 자동으로 해제되므로 특별한 작업 불필요
    chunk.row_ptr.clear();
    chunk.col_idx.clear();
    chunk.labels.clear();
    chunk.local_node_flags.clear();
}

// GPU용 Partition Info 구조체
struct PartitionInfoGPU {
    double P_L;   // DMOLP Penalty
};

// ==================== Warp Reduction Helper ====================
__inline__ __device__ double warpReduceSum(double val) {
    for (int offset = warpSize / 2; offset > 0; offset >>= 1)
        val += __shfl_down_sync(0xffffffff, val, offset);
    return val;
}

__inline__ __device__ int warpReduceMax(int val) {
    for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
        int other = __shfl_down_sync(0xffffffff, val, offset);
        val = max(val, other);
    }
    return val;
}

// ==================== CUDA 커널 (고성능 워프 최적화 - 기본) ====================
/**
 * 고성능 경계 노드 라벨 전파 GPU 커널
 * - 워프 협력적 처리
 * - 벡터화된 메모리 접근
 * - 공유 메모리 뱅크 충돌 최소화
 */
__global__ void boundaryLPKernel_optimized(
    const int* __restrict__ row_ptr, 
    const int* __restrict__ col_idx,
    const int* __restrict__ labels_old, 
    int* __restrict__ labels_new,
    const double* __restrict__ penalty,
    const int* __restrict__ boundary_nodes, 
    int boundary_count,
    int num_partitions,
    int labels_count)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= boundary_count) return;

    int node = boundary_nodes[idx];
    if (node < 0 || node >= labels_count) return;
    int my_label = labels_old[node];

    // 패딩된 공유 메모리 (뱅크 충돌 방지)
    extern __shared__ double shared_mem[];
    double* scores = shared_mem;
    
    // 워프 협력적 초기화
    for (int l = threadIdx.x; l < num_partitions; l += blockDim.x) {
        scores[l] = 0.0;
    }
    __syncthreads();

    // 로컬 카운터 배열 (레지스터 사용)
    double local_counts[32] = {0.0}; // 최대 32개 파티션 지원
    int max_partitions = (num_partitions < 32 ? num_partitions : 32);
    
    // 이웃 노드 순회 및 카운팅
    int start = row_ptr[node];
    int end = row_ptr[node + 1];
    
    for (int e = start; e < end; e++) {
        int neighbor = col_idx[e];
        if (neighbor >= 0 && neighbor < labels_count) {
            int neighbor_label = labels_old[neighbor];
            if (neighbor_label >= 0 && neighbor_label < max_partitions) {
                local_counts[neighbor_label] += 1.0;
            }
        }
    }
    
    // 로컬 카운트를 공유 메모리에 합산 (atomic 최소화)
    for (int l = 0; l < max_partitions; l++) {
        if (local_counts[l] > 0.0) {
            atomicAdd(&scores[l], local_counts[l]);
        }
    }
    __syncthreads();
    
    // 패널티 적용 (워프 협력)
    for (int l = threadIdx.x; l < num_partitions; l += blockDim.x) {
        if (scores[l] > 0.0) {
            scores[l] = scores[l] * (1.0 + penalty[l]);
        }
    }
    __syncthreads();

    // 워프 수준 최대값 찾기
    int best_label = my_label;
    double best_score = (my_label < num_partitions) ? scores[my_label] : 0.0;
    
    // 병렬 스캔으로 최대값 찾기
    for (int l = 0; l < num_partitions; l++) {
        if (scores[l] > best_score) {
            best_score = scores[l];
            best_label = l;
        }
    }
    
    // 결과 저장
    labels_new[node] = best_label;
}

// ==================== CUDA 커널 (안전 버전) ====================
__global__ void boundaryLPKernel_safe(
    const int* __restrict__ row_ptr, 
    const int* __restrict__ col_idx,
    const int* __restrict__ labels_old, 
    int* __restrict__ labels_new,
    const double* __restrict__ penalty,
    const int* __restrict__ boundary_nodes, 
    int boundary_count,
    int num_partitions,
    int labels_count)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= boundary_count) return;

    int node = boundary_nodes[idx];
    if (node < 0 || node >= labels_count) return;
    int my_label = labels_old[node];

    extern __shared__ double scores[];
    for (int l = threadIdx.x; l < num_partitions; l += blockDim.x) {
        scores[l] = 0.0;
    }
    __syncthreads();

    int start = row_ptr[node];
    int end = row_ptr[node + 1];
    for (int e = start; e < end; e++) {
        int neighbor = col_idx[e];
        if (neighbor >= 0 && neighbor < labels_count) {
            int neighbor_label = labels_old[neighbor];
            if (neighbor_label >= 0 && neighbor_label < num_partitions) {
                atomicAdd(&scores[neighbor_label], 1.0);
            }
        }
    }
    __syncthreads();

    for (int l = threadIdx.x; l < num_partitions; l += blockDim.x) {
        if (scores[l] > 0.0) {
            scores[l] = scores[l] * (1.0 + penalty[l]);
        }
    }
    __syncthreads();

    int best_label = my_label;
    double best_score = (my_label >= 0 && my_label < num_partitions) ? scores[my_label] : 0.0;
    for (int l = 0; l < num_partitions; l++) {
        if (scores[l] > best_score) {
            best_score = scores[l];
            best_label = l;
        }
    }
    if (best_label != my_label) {
        labels_new[node] = best_label;
    }
}

// ==================== CUDA 커널 (워프당 1-노드, 핀드 최적화 전용) ====================
__global__ void boundaryLPKernel_memory_optimized_warp(
    const int* __restrict__ row_ptr,
    const int* __restrict__ col_idx,
    const int* __restrict__ labels_old,
    int* __restrict__ labels_new,
    const double* __restrict__ penalty,
    const int* __restrict__ boundary_nodes,
    int boundary_count,
    int num_partitions,
    int labels_count)
{
    const int lane = threadIdx.x & 31;
    const int warp_id_in_block = threadIdx.x >> 5;
    const int warps_per_block = blockDim.x >> 5;

    const int node_idx = blockIdx.x * warps_per_block + warp_id_in_block;
    if (node_idx >= boundary_count) return;

    const int node = boundary_nodes[node_idx];
    if (node < 0 || node >= labels_count) return;
    const int my_label = labels_old[node];

    extern __shared__ double shm[];
    double* scores = shm + warp_id_in_block * num_partitions;

    for (int l = lane; l < num_partitions; l += 32) {
        scores[l] = 0.0;
    }
    __syncwarp();

    const int start = row_ptr[node];
    const int end   = row_ptr[node + 1];
    for (int e = start + lane; e < end; e += 32) {
        const int nei = col_idx[e];
        if (nei >= 0 && nei < labels_count) {
            const int lbl = labels_old[nei];
            if (lbl >= 0 && lbl < num_partitions) {
                atomicAdd(&scores[lbl], 1.0);
            }
        }
    }
    __syncwarp();

    for (int l = lane; l < num_partitions; l += 32) {
        if (scores[l] > 0.0) {
            scores[l] = scores[l] * (1.0 + penalty[l]);
        }
    }
    __syncwarp();

    if (lane == 0) {
        int best_label = my_label;
        double best_score = (my_label >= 0 && my_label < num_partitions) ? scores[my_label] : 0.0;
        for (int l = 0; l < num_partitions; l++) {
            if (scores[l] > best_score) {
                best_score = scores[l];
                best_label = l;
            }
        }
        if (best_label != my_label) {
            labels_new[node] = best_label;
        }
    }
}

// ==================== Public API ====================
/**
 * 고성능 GPU 라벨 전파 함수
 * 최적화된 워프 레벨 병렬성과 메모리 접근 패턴 사용
 */
void runBoundaryLPOnGPU_Optimized(
    const std::vector<int>& row_ptr,
    const std::vector<int>& col_idx,
    const std::vector<int>& labels_old,
    std::vector<int>& labels_new,
    const std::vector<double>& penalty,
    const std::vector<int>& boundary_nodes,
    int num_partitions)
{
    // 성능 측정 (CUDA 이벤트)
    hipEvent_t ev_start, ev_stop; float ms=0.0f; hipEventCreate(&ev_start); hipEventCreate(&ev_stop);
    hipEventRecord(ev_start);
    
    // CUDA 스트림 생성
    hipStream_t stream;
    hipStreamCreate(&stream);
    
    // GPU 메모리 할당
    int* d_row_ptr; int* d_col_idx;
    int* d_labels_old; int* d_labels_new;
    int* d_boundary;
    double* d_penalty;

    size_t row_size = row_ptr.size() * sizeof(int);
    size_t col_size = col_idx.size() * sizeof(int);
    size_t labels_size = labels_old.size() * sizeof(int);
    size_t boundary_size = boundary_nodes.size() * sizeof(int);
    size_t penalty_size = penalty.size() * sizeof(double);

    hipMalloc(&d_row_ptr, row_size);
    hipMalloc(&d_col_idx, col_size);
    hipMalloc(&d_labels_old, labels_size);
    hipMalloc(&d_labels_new, labels_size);
    hipMalloc(&d_boundary, boundary_size);
    hipMalloc(&d_penalty, penalty_size);

    // 비동기 메모리 복사
    hipMemcpyAsync(d_row_ptr, row_ptr.data(), row_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_col_idx, col_idx.data(), col_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_labels_old, labels_old.data(), labels_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_labels_new, labels_new.data(), labels_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_boundary, boundary_nodes.data(), boundary_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_penalty, penalty.data(), penalty_size, hipMemcpyHostToDevice, stream);

    // 최적화된 커널 실행 설정
    int threads = 256;  // 높은 occupancy
    int blocks = (boundary_nodes.size() + threads - 1) / threads;
    size_t shared_mem = (num_partitions + 32) * sizeof(double);  // 뱅크 충돌 방지

    // 커널 실행
    boundaryLPKernel_optimized<<<blocks, threads, shared_mem, stream>>>(
        d_row_ptr, d_col_idx,
        d_labels_old, d_labels_new,
        d_penalty,
        d_boundary, boundary_nodes.size(),
        num_partitions,
        static_cast<int>(labels_old.size()));

    // 결과 복사
    hipMemcpyAsync(labels_new.data(), d_labels_new, labels_size, 
                    hipMemcpyDeviceToHost, stream);
    
    // 동기화 및 시간 측정 종료
    hipStreamSynchronize(stream);
    hipEventRecord(ev_stop); hipEventSynchronize(ev_stop); hipEventElapsedTime(&ms, ev_start, ev_stop);
    
    printf("[GPU-Optimized] Execution time: %ld μs (boundary nodes: %zu)\n", 
           (long)(ms*1000.0f), boundary_nodes.size());
    hipEventDestroy(ev_start); hipEventDestroy(ev_stop);

    // 정리
    hipStreamDestroy(stream);
    hipFree(d_row_ptr);
    hipFree(d_col_idx);
    hipFree(d_labels_old);
    hipFree(d_labels_new);
    hipFree(d_boundary);
    hipFree(d_penalty);
}

// 안전한 GPU 라벨 전파 (fallback)
void runBoundaryLPOnGPU_Safe(
    const std::vector<int>& row_ptr,
    const std::vector<int>& col_idx,
    const std::vector<int>& labels_old,
    std::vector<int>& labels_new,
    const std::vector<double>& penalty,
    const std::vector<int>& boundary_nodes,
    int num_partitions)
{
    hipEvent_t ev_start, ev_stop; float ms=0.0f; hipEventCreate(&ev_start); hipEventCreate(&ev_stop);
    hipEventRecord(ev_start);
    hipStream_t stream; hipStreamCreate(&stream);

    int *d_row_ptr, *d_col_idx, *d_labels_old, *d_labels_new, *d_boundary; double* d_penalty;
    size_t row_size = row_ptr.size() * sizeof(int);
    size_t col_size = col_idx.size() * sizeof(int);
    size_t labels_size = labels_old.size() * sizeof(int);
    size_t boundary_size = boundary_nodes.size() * sizeof(int);
    size_t penalty_size = penalty.size() * sizeof(double);

    hipMalloc(&d_row_ptr, row_size);
    hipMalloc(&d_col_idx, col_size);
    hipMalloc(&d_labels_old, labels_size);
    hipMalloc(&d_labels_new, labels_size);
    hipMalloc(&d_boundary, boundary_size);
    hipMalloc(&d_penalty, penalty_size);

    hipMemcpyAsync(d_row_ptr, row_ptr.data(), row_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_col_idx, col_idx.data(), col_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_labels_old, labels_old.data(), labels_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_labels_new, labels_new.data(), labels_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_boundary, boundary_nodes.data(), boundary_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_penalty, penalty.data(), penalty_size, hipMemcpyHostToDevice, stream);

    int threads = 256;
    int blocks = (boundary_nodes.size() + threads - 1) / threads;
    size_t shared_mem = (num_partitions + 8) * sizeof(double);

    boundaryLPKernel_safe<<<blocks, threads, shared_mem, stream>>>(
        d_row_ptr, d_col_idx, d_labels_old, d_labels_new, d_penalty, d_boundary,
        boundary_nodes.size(), num_partitions,
        static_cast<int>(labels_old.size()));

    hipMemcpyAsync(labels_new.data(), d_labels_new, labels_size, hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    hipEventRecord(ev_stop); hipEventSynchronize(ev_stop); hipEventElapsedTime(&ms, ev_start, ev_stop);
    printf("[GPU-Safe] Execution time: %ld μs (boundary nodes: %zu)\n", (long)(ms*1000.0f), boundary_nodes.size());
    hipEventDestroy(ev_start); hipEventDestroy(ev_stop);

    hipStreamDestroy(stream);
    hipFree(d_row_ptr); hipFree(d_col_idx); hipFree(d_labels_old);
    hipFree(d_labels_new); hipFree(d_boundary); hipFree(d_penalty);
}

// 핀드 메모리 최적화 경계 LP (워프당 1-노드)
void runBoundaryLPOnGPU_PinnedOptimized(
    const std::vector<int>& row_ptr,
    const std::vector<int>& col_idx,
    const std::vector<int>& labels_old,
    std::vector<int>& labels_new,
    const std::vector<double>& penalty,
    const std::vector<int>& boundary_nodes,
    int num_partitions)
{
    GPUMemoryManager::enableLeakDetection();
    hipEvent_t ev_start, ev_stop; float ms=0.0f; hipEventCreate(&ev_start); hipEventCreate(&ev_stop);
    hipEventRecord(ev_start);
    
    try {
        hipStream_t stream; hipStreamCreate(&stream);
        
        size_t row_size = row_ptr.size() * sizeof(int);
        size_t col_size = col_idx.size() * sizeof(int);
        size_t labels_size = labels_old.size() * sizeof(int);
        size_t boundary_size = boundary_nodes.size() * sizeof(int);
        size_t penalty_size = penalty.size() * sizeof(double);
        
        auto row_buffer = PinnedMemoryPool::acquireBuffer(row_size);
        auto col_buffer = PinnedMemoryPool::acquireBuffer(col_size);
        auto labels_old_buffer = PinnedMemoryPool::acquireBuffer(labels_size);
        auto labels_new_buffer = PinnedMemoryPool::acquireBuffer(labels_size);
        auto boundary_buffer = PinnedMemoryPool::acquireBuffer(boundary_size);
        auto penalty_buffer = PinnedMemoryPool::acquireBuffer(penalty_size);
        
        memcpy(row_buffer->host_ptr, row_ptr.data(), row_size);
        memcpy(col_buffer->host_ptr, col_idx.data(), col_size);
        memcpy(labels_old_buffer->host_ptr, labels_old.data(), labels_size);
        memcpy(labels_new_buffer->host_ptr, labels_new.data(), labels_size);
        memcpy(boundary_buffer->host_ptr, boundary_nodes.data(), boundary_size);
        memcpy(penalty_buffer->host_ptr, penalty.data(), penalty_size);
        
        hipMemcpyAsync(row_buffer->device_ptr, row_buffer->host_ptr, row_size, hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(col_buffer->device_ptr, col_buffer->host_ptr, col_size, hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(labels_old_buffer->device_ptr, labels_old_buffer->host_ptr, labels_size, hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(labels_new_buffer->device_ptr, labels_new_buffer->host_ptr, labels_size, hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(boundary_buffer->device_ptr, boundary_buffer->host_ptr, boundary_size, hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(penalty_buffer->device_ptr, penalty_buffer->host_ptr, penalty_size, hipMemcpyHostToDevice, stream);
        
        int warpsPerBlock = 8;
        int blockSize = warpsPerBlock * 32;
        int gridSize = (static_cast<int>(boundary_nodes.size()) + warpsPerBlock - 1) / warpsPerBlock;
        size_t shared_mem = static_cast<size_t>(num_partitions) * warpsPerBlock * sizeof(double);

        boundaryLPKernel_memory_optimized_warp<<<gridSize, blockSize, shared_mem, stream>>>(
            (int*)row_buffer->device_ptr, (int*)col_buffer->device_ptr,
            (int*)labels_old_buffer->device_ptr, (int*)labels_new_buffer->device_ptr,
            (double*)penalty_buffer->device_ptr,
            (int*)boundary_buffer->device_ptr, boundary_nodes.size(),
            num_partitions,
            static_cast<int>(labels_old.size()));
        
        hipMemcpyAsync(labels_new_buffer->host_ptr, labels_new_buffer->device_ptr, labels_size, hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
        memcpy(labels_new.data(), labels_new_buffer->host_ptr, labels_size);
        
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("[GPU-Pinned-Error] CUDA error: %s\n", hipGetErrorString(error));
        }
        
        hipStreamDestroy(stream);
        PinnedMemoryPool::releaseBuffer(row_buffer);
        PinnedMemoryPool::releaseBuffer(col_buffer);
        PinnedMemoryPool::releaseBuffer(labels_old_buffer);
        PinnedMemoryPool::releaseBuffer(labels_new_buffer);
        PinnedMemoryPool::releaseBuffer(boundary_buffer);
        PinnedMemoryPool::releaseBuffer(penalty_buffer);
        
    } catch (const std::exception& e) {
        printf("[GPU-Pinned-Exception] %s\n", e.what());
    }
    
    hipEventRecord(ev_stop); hipEventSynchronize(ev_stop); hipEventElapsedTime(&ms, ev_start, ev_stop);
    printf("[GPU-Pinned-Optimized] Execution time: %ld μs (boundary nodes: %zu)\n", (long)(ms*1000.0f), boundary_nodes.size());
    hipEventDestroy(ev_start); hipEventDestroy(ev_stop);
    printf("[GPU-Pinned-Pool] Pool size: %zu buffers\n", PinnedMemoryPool::getPoolSize());
    
}

// 리소스 정리
void cleanupGPUResources() {
    printf("[GPU-Cleanup] Cleaning up all GPU resources...\n");
    PinnedMemoryPool::clearPool();
    GPUMemoryManager::reportLeaks();
    printf("[GPU-Cleanup] Cleanup complete.\n");
}